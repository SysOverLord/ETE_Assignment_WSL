#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>

//#include<windows.h>
//for linux
#include<unistd.h>
#define TEAM_SIZE 400


struct Runner {
	int dist;
	int vel;
};

struct Team {
	Runner* runners;
	int curRunner;
	int id;
};


hipError_t createTeamsWithCuda(Team* teams, Runner* runners, const int size);

//hipError_t simulateRaceWithCuda(Team* teams, Runner* runners, int* finished_team_count, int* placements, const int size);
hipError_t simulateRaceWithCuda(Team* teams, Runner* runners, int* finishedTeamCount, int* placements, int* consoleTeams, const int consoleSize, const int size);


__global__ void createTeamKernel(Team* teams, Runner* runners)
{
	int i = threadIdx.x;
	//RUN_LIMIT
	int size = 4;
	teams[i].runners = &runners[i * size];
	for (int j = 0; j < size; j++)
	{
		teams[i].runners[j].dist = j * 100;
		//At first everybody is at stop.
		teams[i].runners[j].vel = 0;
	}
	teams[i].id = i + 1;
	teams[i].curRunner = 0;

}



__global__ void simulateRaceKernel(Team* teams, Runner* runners, int* finished_team_count, int* placements, int rand_seed)
{
	int i = threadIdx.x;
	teams[i].runners = &runners[i * 4];
	int* curRunner = &teams[i].curRunner;

	if (*curRunner == 4)
	{
		//This team has ended the race.
		return;
	}

	teams[i].runners[*curRunner].dist += teams[i].runners[*curRunner].vel;
	

	if (teams[i].runners[*curRunner].dist >= (*curRunner + 1) * 100)
	{
		teams[i].runners[*curRunner].vel = 0;
		*curRunner += 1;
	}

	if (*curRunner == 4)
	{

		int place_index = atomicAdd(finished_team_count, 1);
		placements[place_index] = i;
		if (place_index == 0)
		{
			printf("\nFirst team to arrive finish line is Team %d\n", i + 1);
			printf("-----------------------------------------------\n");
			for (int j = 0; j < 4; j++)
			{
				printf("Team %d Runner %d VEL:%d DIST:%d\n", i + 1, j + 1, teams[i].runners[j].vel, teams[i].runners[j].dist);
			}
			//printf("-----------------------------------------------\n");

			for (int k = 0; k < TEAM_SIZE; k++)
			{
				int tmpRunner = teams[k].curRunner;
				if (tmpRunner == 4)
				{
					tmpRunner -= 1;
				}

				printf("-----------------------------------------------\n");
				printf("Team %d Current Runner %d VEL:%d DIST:%d\n", k + 1, tmpRunner + 1, teams[k].runners[tmpRunner].vel, teams[k].runners[tmpRunner].dist);
			}
		}
	}


	else
	{
		//Race has not ended for this team.
		//Give new velocity to calculate next distance.
		hiprandState_t state;
		hiprand_init(rand_seed, i, 0, &state);

		teams[i].runners[*curRunner].vel = ceil(hiprand_uniform(&state) * 5);
	}
}





int main(int argc, char* argv[])
{

	//pointer for objects
	Team* teams = new Team[TEAM_SIZE];
	Runner* runners = new Runner[TEAM_SIZE * 4];
	//Pointer for placements
	int* placements = new int[TEAM_SIZE];


	hipError_t cudaStatus = createTeamsWithCuda(teams, runners, TEAM_SIZE);

	//Pointers must be reassigned because 
	//Pointer values on the objects are for gpu memory (video-ram)
	//They are needed to be repointed to cpu memory (ram or virtual ram)
	for (int i = 0; i < TEAM_SIZE; i++) {
		teams[i].runners = &runners[i * 4];
	}


	int* consoleTeams;
	int consoleSize = 0;
	int finished_team_count = 0;

	if (argc <= 1)
	{
		consoleTeams = new int[TEAM_SIZE];
		printf("No arguments were passed while running the program.\nPlease state which teams will be shown on the console.\n");
		printf("All numbers must be seperated by space\n");
		do {
			scanf("%d", &consoleTeams[consoleSize++]);
		} while (getchar() != '\n' && consoleSize < TEAM_SIZE);
	}
	else
	{
		consoleTeams = new int[argc - 1];
		for (int i = 1; i < argc; i++)
		{
			sscanf(argv[i], "%d", &consoleTeams[consoleSize++]);
		}
	}
	for (int i = 0; i < consoleSize; i++)
	{
		if (consoleTeams[i] <= 0)
		{
			printf("Can't give an argument below or equal to 0 or NaN. Teams start at 1.");
			exit(-1);
		}
		else if (consoleTeams[i] > TEAM_SIZE)
		{
			printf("Can't select non existent team");
			exit(-2);
		}

	}

	simulateRaceWithCuda(teams, runners, &finished_team_count, placements, consoleTeams, consoleSize, TEAM_SIZE);


	printf("Race has ended The Results are\n");;

	for (int i = 0; i < TEAM_SIZE; i++)
	{
		printf("%d PLACE: TEAM %d\n", i + 1, teams[placements[i]].id);
	}



	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	free(teams);
	free(runners);
	free(placements);
	free(consoleTeams);
	return 0;
}

hipError_t simulateRaceWithCuda(Team* teams, Runner* runners, int* finishedTeamCount, int* placements, int* consoleTeams, const int consoleSize, const int size)
{
	Team* dev_teams;
	Runner* dev_runners;
	int* dev_placements;
	int* dev_count;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	//Memory allocation and Memory copy for host to device.
	cudaStatus = hipMalloc((void**)&dev_placements, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(dev_placements, placements, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}



	cudaStatus = hipMalloc((void**)&dev_teams, size * sizeof(Team));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(dev_teams, teams, size * sizeof(Team), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_runners, size * sizeof(Runner) * 4);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(dev_runners, runners, size * sizeof(Runner) * 4, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_count, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(dev_count, finishedTeamCount, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//End of Memory Allocation and Memory Copy

	//Start race loop
	while (*finishedTeamCount < TEAM_SIZE) {

		srand(time(NULL));
		int rand_seed = rand() % 500 + 1000;

		// Launch a kernel on the GPU with one thread for each element.
		simulateRaceKernel <<<1, size >>> (dev_teams, dev_runners, dev_count, dev_placements, rand_seed);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "simulateRaceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching simulateRaceKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vectors from GPU buffer to host memory.

		cudaStatus = hipMemcpy(teams, dev_teams, size * sizeof(Team), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy device to host on Teams failed!");
			goto Error;
		}


		cudaStatus = hipMemcpy(runners, dev_runners, size * sizeof(Runner) * 4, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy device to host on Runners failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(finishedTeamCount, dev_count, sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy device to host on count failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(placements, dev_placements, size * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy device to host on placements failed!");
			goto Error;
		}

		//Pointers must be reassigned. 
		//Pointer values on the objects are for gpu memory (video-ram).
		//They are needed to be repointed to cpu memory (ram or virtual ram).
		for (int i = 0; i < TEAM_SIZE; i++)
		{
			teams[i].runners = &runners[i * 4];
		}

		for (int i = 0; i < consoleSize; i++)
		{
			printf("-------------------------\n");
			int outTeam = consoleTeams[i];
			for (int j = 0; j < 4; j++)
			{
				printf("Team %d Runner %d VEL:%d DIST:%d\n", outTeam, j + 1, teams[outTeam - 1].runners[j].vel, teams[outTeam - 1].runners[j].dist);
			}
		}

		printf("-------------------------\n");
		printf("Finished Team Count:%d\n", *finishedTeamCount);
		printf("|||||||||||||||||||||||||\n");

		//Sleep function in windows is in milliseconds
		//1000 olucak
		//Sleep(1 * 1000);
		//For linux based
		// It is in seconds for linux.
		sleep(1);

	}
	//End race loop

Error:
	hipFree(dev_teams);
	hipFree(dev_runners);
	hipFree(dev_count);
	hipFree(dev_placements);

	return cudaStatus;

}


hipError_t createTeamsWithCuda(Team* teams, Runner* runners, const int size)
{
	Team* dev_teams;
	Runner* dev_runners;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_teams, size * sizeof(Team));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_teams, teams, size * sizeof(Team), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_runners, size * sizeof(Runner) * 4);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_runners, runners, size * sizeof(Runner) * 4, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	// Launch a kernel on the GPU with one thread for each element.
	createTeamKernel << <1, size >> > (dev_teams, dev_runners);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "createTeamKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//printf("%d", dev_teams[0].runners[0].dist);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching createTeamKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.

	cudaStatus = hipMemcpy(teams, dev_teams, size * sizeof(Team), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy on Teams failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(runners, dev_runners, size * sizeof(Runner) * 4, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy on Runners failed!");
		goto Error;
	}




Error:
	hipFree(dev_teams);
	hipFree(dev_runners);

	return cudaStatus;

}